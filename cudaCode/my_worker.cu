#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "my_worker.cuh"
namespace tjn{
  __device__ float *values_d;
  __device__ float *deltas_d;
  __device__ unsigned int start_d;
  __device__ unsigned int end_d;
  __device__ unsigned int *oeoffset_d;
  __device__ unsigned int *cur_oeoff_d;
  __device__ unsigned int *size_oe_d;
  __device__ char *node_type_d;
  __device__ float *spnode_datas_d;
  __device__ float *bound_node_values_d;
  /**
   * @brief node type
  */
  enum NodeType {
    SingleNode = 0,
    OnlyOutNode = 1,
    OnlyInNode = 2,
    BothOutInNode = 3,
    InnerNode = 4,
    OutMaster = 5,
    BothOutInMaster = 6,
  };
  void value2last(float *last_values_d,float *values_d,unsigned int *v_d,int size){
    dim3 block(512);
    dim3 grid((size-1)/block.x + 1);
    value2last_real<<<grid,block>>>(last_values_d,values_d,v_d,size);
  }

  __global__
  void value2last_real(float *last_values_d,float *values_d,unsigned int *v_d,int size){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < size){
      last_values_d[v_d[index]] = values_d[v_d[index]];
    }
    // int stride = gridDim.x * blockDim.x;
    // for(int i=index;i<size;i+=stride){
    //     last_values_d[ v_d[i] ] = values_d[ v_d[i] ];
    //     // printf("lastvalueis %f\n", last_values_d[v_d[i]]);
    //     // printf("numis %d\n", size);
    // }
  }
  void init(float *spnode_datas_d, float *bound_node_values_d, float *deltas_d, float *values_d, 
            unsigned int *oeoffset_d, unsigned int *iboffset_d, unsigned int *isoffset_d, 
            unsigned int *size_oe_d, unsigned int *size_ib_d, unsigned int *size_is_d, 
            unsigned int start_d, unsigned int end_d, 
            unsigned int *cur_oeoff_d, unsigned int *cur_iboff_d, unsigned int *cur_isoff_d, 
            char *node_type_d){

      init_real<<<1,1>>>(spnode_datas_d, bound_node_values_d, deltas_d, values_d, 
                      oeoffset_d, iboffset_d, isoffset_d, 
                      size_oe_d, size_ib_d, size_is_d, 
                      start_d, end_d, 
                      cur_oeoff_d, cur_iboff_d, cur_isoff_d, 
                      node_type_d);

  }

  __global__
  void init_real(float *spnode_datas_d, float *bound_node_values_d, float *deltas_d, float *values_d, 
            unsigned int *oeoffset_d, unsigned int *iboffset_d, unsigned int *isoffset_d, 
            unsigned int *size_oe_d, unsigned int *size_ib_d, unsigned int *size_is_d, 
            unsigned int start_d, unsigned int end_d, 
            unsigned int *cur_oeoff_d, unsigned int *cur_iboff_d, unsigned int *cur_isoff_d, 
            char *node_type_d){

        tjn::values_d = values_d;
        tjn::deltas_d = deltas_d;
        tjn::start_d = start_d;
        tjn::end_d = end_d;
        tjn::oeoffset_d = oeoffset_d;
        tjn::size_oe_d = size_oe_d;
        tjn::cur_oeoff_d = cur_oeoff_d;
        tjn::node_type_d = node_type_d;
        tjn::spnode_datas_d = spnode_datas_d;
        tjn::bound_node_values_d = bound_node_values_d;

  }
  void g_function_pr(unsigned int start_d, unsigned int end_d){
    dim3 block(512);
    dim3 grid((end_d - start_d - 1) / block.x + 1);
    g_function_pr_real<<<grid, block>>>();
  }

    __global__
  void g_function_pr_real(){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < end_d - start_d){
      // float delta = atomicExch(&deltas_d[index], 0);
      if(isChange_pr(deltas_d[index], end_d - start_d)){
        pr_Ingress(index);
        // atomicAdd(&values_d[index], deltas_d[index]);
        // // values_d[index] += deltas_d[index];
        // // __syncthreads();
        // // __threadfence();
        // float delta = atomicExch(&deltas_d[index], 0);
        // // __syncthreads();
        // // __threadfence();
        // // atomicAdd(&values_d[index], delta);
        // // atomicAdd(&values_d[index], delta);
        // // values_d[index] += delta;
        // // values_d[index] += delta;
        // unsigned int out_degree = max(size_oe_d[index],1);
        // // atomicExch(&deltas_d[index], 0);
        // // __syncthreads();
        // // __threadfence();
        // float outv = delta * 0.85f / out_degree;
        // for(unsigned int i=cur_oeoff_d[index];i<cur_oeoff_d[index] + size_oe_d[index];i++){
        //   // deltas_d[ oeoffset_d[i] ] += outv;
        //   atomicAdd(&deltas_d[oeoffset_d[i]],outv);
        // } 
        // // __syncthreads();
        // // __threadfence();
          
        
      }else{
        return ;
      }
      

    }else{
      return ;
    }
        // int row = threadIdx.y;//innervertices'order
    // int col = threadIdx.x;
    // if(index < offsize){
    //   if(isChange_pr(deltas_d[start_d + curId_d[index]], end_d - start_d)){
    //     double delta = deltas_d[start_d + curId_d[index]];
    //     unsigned int out_degree = size_oe_d[start_d + curId_d[index]];
    //     double outv = delta * 0.85f / out_degree;
    //     // __syncthreads();
    //     // atomicAdd(&deltas_d[oeoffset_d[index]], outv);
    //     __syncthreads();
    //     __threadfence();
    //     deltas_d[curId_d[index]] = 0;
    //     __syncthreads();
    //     __threadfence();
    //     deltas_d[oeoffset_d[index]] += outv;
    //     if( index == 0 || curId_d[index] != curId_d[index-1]){
    //       // printf("curID is %d",curId_d[index]);
    //       // deltas_d[curId_d[index]] -= delta;
    //       __syncthreads();
    //     __threadfence();
    //       values_d[curId_d[index]] += deltas_d[curId_d[index]];
    //       // atomicAdd(&deltas_d[start_d + curId_d[index]], -delta);
    //       // atomicAdd(&values_d[start_d + curId_d[index]], deltas_d[start_d + curId_d[index]]);
    //     }

    //     __syncthreads();
    //     __threadfence();
    //   }
    // }
    // // if(oeoffset_d[row * blockDim.x + col] == end_d - start_d){
    //   if(size_oe_d[start_d + row] == 0)deltas_d[row] = 0;
    //   return ;
    // }
    // if(isChange_pr(deltas_d[start_d + row], end_d - start_d)){

    //   float delta = deltas_d[start_d + row];

    //   unsigned int out_degree = size_oe_d[start_d + row];
    //   float outv = delta * 0.85f / out_degree;
    //   __syncthreads();
    //   atomicAdd(&deltas_d[oeoffset_d[row * blockDim.x + col]], outv);
    //   if( (row*blockDim.x + col) % blockDim.x == 0){
    //     atomicAdd(&deltas_d[start_d + row], -delta);
    //     atomicAdd(&values_d[start_d + row], deltas_d[start_d + row]);
    //   }
    // }
  }

  void g_function_compr(unsigned int start_d, unsigned int end_d){
    dim3 block(512);
    dim3 grid((end_d - start_d - 1) / block.x + 1);
    g_function_compr_real<<<grid, block>>>();
  }

  __global__
  void g_function_compr_real(){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < end_d - start_d){
      if(isChange_pr(deltas_d[index], end_d - start_d)){
        switch(node_type_d[index]){
          case NodeType::SingleNode:
            {
              pr_singleNode(index);
            }
            break;
          case NodeType::OnlyInNode:
            {

            }
            break;
          case NodeType::OnlyOutNode:
            {

            }
            break;
          case NodeType::BothOutInNode:
            {

            }
            {

            }
            break;
          case NodeType::OutMaster:
            {

            }
            break;
          case NodeType::BothOutInMaster:
            {

            }
            {

            }
            break;
        }
      }else{
        return ;
      }
    }else{
      return ;
    }
  }

  __device__
  bool isChange_pr(float delta, int verticesNum){
    if (fabs(delta) > (1e-6) / verticesNum) {
      return true;
    } else {
      return false;
    }
  }

  __device__
  inline void pr_Ingress(int index){

    // atomicAdd(&values_d[index], deltas_d[index]);//加在此处也可

    float delta = atomicExch(&deltas_d[index], 0);

    unsigned int out_degree = max(size_oe_d[index],1);

    float outv = delta * 0.85f / out_degree;

    for(unsigned int i=cur_oeoff_d[index];i<cur_oeoff_d[index] + size_oe_d[index];i++){

      atomicAdd(&deltas_d[oeoffset_d[i]],outv);

    } 

    atomicAdd(&values_d[index], delta);

  }


  __device__
  inline void pr_singleNode(int index){

    // atomicAdd(&values_d[index], deltas_d[index]);//加在此处也可

    float delta = atomicExch(&deltas_d[index], 0);

    unsigned int out_degree = max(size_oe_d[index],1);

    float outv = delta * 0.85f / out_degree;

    for(unsigned int i=cur_oeoff_d[index];i<cur_oeoff_d[index] + size_oe_d[index];i++){

      atomicAdd(&deltas_d[oeoffset_d[i]],outv);

    } 

    atomicAdd(&values_d[index], delta);

  }

  __device__
  inline void pr_onlyInNode(int index){

    float delta = atomicExch(&deltas_d[index], 0);
    
    unsigned int out_degree = max(size_oe_d[index],1);

    float outv = delta * 0.85f / out_degree;


  }

  __device__
  inline void pr_onlyOutNode(int index){

  }

  __device__
  inline void pr_bothOutInNode(int index){

  }

  __device__
  inline void pr_outMaster(int index){

  }

  __device__
  inline void pr_bothOutInMaster(int index){

  }
}
