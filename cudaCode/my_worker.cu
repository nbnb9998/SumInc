#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "my_worker.cuh"
#include "grape/utils/vertex_array.h"
namespace tjn{
  __device__ float *values_d;
  __device__ float *deltas_d;
  __device__ unsigned int start_d;
  __device__ unsigned int end_d;
  __device__ unsigned int *oeoffset_d;
  __device__ unsigned int *curOff_d;
  __device__ unsigned int *size_d;
  __device__ char *node_type_d;
  /**
   * @brief node type
  */
  enum NodeType {
    SingleNode = 0,
    OnlyOutNode = 1,
    OnlyInNode = 2,
    BothOutInNode = 3,
    InnerNode = 4,
    OutMaster = 5,
    BothOutInMaster = 6,
  };
  void value2last(float *last_values_d,float *values_d,unsigned int *v_d,int size){
    dim3 block(512);
    dim3 grid((size-1)/block.x + 1);
    value2last_real<<<grid,block>>>(last_values_d,values_d,v_d,size);
  }

  __global__
  void value2last_real(float *last_values_d,float *values_d,unsigned int *v_d,int size){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < size){
      last_values_d[v_d[index]] = values_d[v_d[index]];
    }
    // int stride = gridDim.x * blockDim.x;
    // for(int i=index;i<size;i+=stride){
    //     last_values_d[ v_d[i] ] = values_d[ v_d[i] ];
    //     // printf("lastvalueis %f\n", last_values_d[v_d[i]]);
    //     // printf("numis %d\n", size);
    // }
  }
  void init(float *deltas_d, float *values_d, unsigned int *oeoffset_d, unsigned int *size_d, unsigned int start_d, unsigned int end_d, unsigned int *curOff_d, char *node_type_d){
    init_real<<<1,1>>>(deltas_d, values_d, oeoffset_d, size_d, start_d, end_d, curOff_d, node_type_d);
  }
  __global__
  void init_real(float *deltas_d, float *values_d, unsigned int *oeoffset_d, unsigned int *size_d, unsigned int start_d, unsigned int end_d, unsigned int *curOff_d, char *node_type_d){
    tjn::values_d = values_d;
    tjn::deltas_d = deltas_d;
    tjn::start_d = start_d;
    tjn::end_d = end_d;
    tjn::oeoffset_d = oeoffset_d;
    tjn::size_d = size_d;
    tjn::curOff_d = curOff_d;
    tjn::node_type_d = node_type_d;
  }
  void g_function_pr(unsigned int start_d, unsigned int end_d){
    dim3 block(512);
    dim3 grid((end_d - start_d - 1) / block.x + 1);
    g_function_pr_real<<<grid, block>>>();
  }

    __global__
  void g_function_pr_real(){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < end_d - start_d){
      // float delta = atomicExch(&deltas_d[index], 0);
      if(isChange_pr(deltas_d[index], end_d - start_d)){
        atomicAdd(&values_d[index], deltas_d[index]);
        // values_d[index] += deltas_d[index];
        // __syncthreads();
        // __threadfence();
        float delta = atomicExch(&deltas_d[index], 0);
        // __syncthreads();
        // __threadfence();
        // atomicAdd(&values_d[index], delta);
        // atomicAdd(&values_d[index], delta);
        // values_d[index] += delta;
        // values_d[index] += delta;
        unsigned int out_degree = max(size_d[index],1);
        // atomicExch(&deltas_d[index], 0);
        // __syncthreads();
        // __threadfence();
          float outv = delta * 0.85f / out_degree;
          for(unsigned int i=curOff_d[index];i<curOff_d[index] + size_d[index];i++){
            // deltas_d[ oeoffset_d[i] ] += outv;
            atomicAdd(&deltas_d[oeoffset_d[i]],outv);
          } 
          // __syncthreads();
          // __threadfence();
          
        
      }else{
        return ;
      }
      

    }else{
      return ;
    }
        // int row = threadIdx.y;//innervertices'order
    // int col = threadIdx.x;
    // if(index < offsize){
    //   if(isChange_pr(deltas_d[start_d + curId_d[index]], end_d - start_d)){
    //     double delta = deltas_d[start_d + curId_d[index]];
    //     unsigned int out_degree = size_d[start_d + curId_d[index]];
    //     double outv = delta * 0.85f / out_degree;
    //     // __syncthreads();
    //     // atomicAdd(&deltas_d[oeoffset_d[index]], outv);
    //     __syncthreads();
    //     __threadfence();
    //     deltas_d[curId_d[index]] = 0;
    //     __syncthreads();
    //     __threadfence();
    //     deltas_d[oeoffset_d[index]] += outv;
    //     if( index == 0 || curId_d[index] != curId_d[index-1]){
    //       // printf("curID is %d",curId_d[index]);
    //       // deltas_d[curId_d[index]] -= delta;
    //       __syncthreads();
    //     __threadfence();
    //       values_d[curId_d[index]] += deltas_d[curId_d[index]];
    //       // atomicAdd(&deltas_d[start_d + curId_d[index]], -delta);
    //       // atomicAdd(&values_d[start_d + curId_d[index]], deltas_d[start_d + curId_d[index]]);
    //     }

    //     __syncthreads();
    //     __threadfence();
    //   }
    // }
    // // if(oeoffset_d[row * blockDim.x + col] == end_d - start_d){
    //   if(size_d[start_d + row] == 0)deltas_d[row] = 0;
    //   return ;
    // }
    // if(isChange_pr(deltas_d[start_d + row], end_d - start_d)){

    //   float delta = deltas_d[start_d + row];

    //   unsigned int out_degree = size_d[start_d + row];
    //   float outv = delta * 0.85f / out_degree;
    //   __syncthreads();
    //   atomicAdd(&deltas_d[oeoffset_d[row * blockDim.x + col]], outv);
    //   if( (row*blockDim.x + col) % blockDim.x == 0){
    //     atomicAdd(&deltas_d[start_d + row], -delta);
    //     atomicAdd(&values_d[start_d + row], deltas_d[start_d + row]);
    //   }
    // }
  }

  void g_function_compr(unsigned int start_d, unsigned int end_d){
    dim3 block(512);
    dim3 grid((end_d - start_d - 1) / block.x + 1);
    g_function_compr_real<<<grid, block>>>();
  }

  __global__
  void g_function_compr_real(){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < end_d - start_d){
      switch(node_type_d[index]){
        case NodeType::SingleNode:
          {
            
          }
          break;
        case NodeType::OnlyInNode:
          {

          }
          break;
        case NodeType::OnlyOutNode:
          {

          }
          break;
        case NodeType::BothOutInNode:
          {

          }
          {

          }
          break;
        case NodeType::OutMaster:
          {

          }
          break;
        case NodeType::BothOutInMaster:
          {

          }
          {

          }
          break;
      }
    }else{
      return ;
    }
  }

  __device__
  bool isChange_pr(float delta, int verticesNum){
    if (fabs(delta) > (1e-6) / verticesNum) {
      return true;
    } else {
      return false;
    }
  }

}
