#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "my_ssspworker.cuh"
// #include "freshman.h"
namespace tjnsssp{
    __device__ unsigned int *oeoffset_d;
    __device__ unsigned int *iboffset_d;
    __device__ unsigned int *isoffset_d;

    __device__ unsigned int *cur_oeoff_d;
    __device__ unsigned int *cur_iboff_d;
    __device__ unsigned int *cur_isoff_d;

    __device__ unsigned int *size_oe_d;
    __device__ unsigned int *size_ib_d;
    __device__ unsigned int *size_is_d;

    __device__ int *oe_edata_d;
    __device__ int *ib_edata_d;
    __device__ int *is_edata_d;

    __device__ int *deltas_d;
    __device__ int *values_d;

    __device__ unsigned int *cur_modified_d;
    __device__ unsigned int *cur_modified_size_d;//长度为1,只记录当前修改队列的长度
    // __device__ unsigned int *next_modified_d;
    // __device__ unsigned int *next_modified_size_d;//长度为num,记录每个modified顶点产生的新的modified顶点数量,这里设为num的作用是使得每个线程能正确处理当前顶点产生的新modified
    __device__ unsigned int next_modified_allsize_d;
    __device__ unsigned int *is_modified_d;//记录顶点是否被修改
    __device__ unsigned int num;

    __device__ char *node_type_d;

    __device__ int *sem;
    __device__ int curpos = 0;
    
    void init(unsigned int *oeoffset_d, int *oe_edata_d, unsigned int *cur_oeoff_d, int *deltas_d, int *values_d, unsigned int *size_oe_d, int FLAGS_sssp_source, 
              unsigned int *cur_modified_d, unsigned int *cur_modified_size_d, unsigned int *is_modified_d, unsigned int num, 
              unsigned int *iboffset_d, int *ib_edata_d, unsigned int *cur_iboff_d, unsigned int *size_ib_d, 
              unsigned int *isoffset_d, int *is_edata_d, unsigned int *cur_isoff_d, unsigned int *size_is_d, 
              char *node_type_d){

                int *sem_d;
                hipMalloc(&sem_d, sizeof(int) *1);

        init_real<<<1,1>>>(oeoffset_d, oe_edata_d, cur_oeoff_d, deltas_d, values_d, size_oe_d, FLAGS_sssp_source, 
                           cur_modified_d, cur_modified_size_d, is_modified_d, num, 
                           iboffset_d, ib_edata_d, cur_iboff_d, size_ib_d, 
                           isoffset_d, is_edata_d, cur_isoff_d, size_is_d, 
                           node_type_d, sem_d);

    }

    __global__
    void init_real(unsigned int *oeoffset_d, int *oe_edata_d, unsigned int *cur_oeoff_d, int *deltas_d, int *values_d, unsigned int *size_oe_d, int FLAGS_sssp_source, 
              unsigned int *cur_modified_d, unsigned int *cur_modified_size_d, unsigned int *is_modified_d, unsigned int num, 
              unsigned int *iboffset_d, int *ib_edata_d, unsigned int *cur_iboff_d, unsigned int *size_ib_d, 
              unsigned int *isoffset_d, int *is_edata_d, unsigned int *cur_isoff_d, unsigned int *size_is_d, 
              char *node_type_d, int *sem_d){

        tjnsssp::oeoffset_d = oeoffset_d;
        tjnsssp::iboffset_d = iboffset_d;
        tjnsssp::isoffset_d = isoffset_d;

        tjnsssp::cur_oeoff_d = cur_oeoff_d;
        tjnsssp::cur_iboff_d = cur_iboff_d;
        tjnsssp::cur_isoff_d = cur_isoff_d;

        tjnsssp::size_oe_d = size_oe_d;
        tjnsssp::size_ib_d = size_ib_d;
        tjnsssp::size_is_d = size_is_d;

        tjnsssp::oe_edata_d = oe_edata_d;
        tjnsssp::ib_edata_d = ib_edata_d;

        tjnsssp::deltas_d = deltas_d;
        tjnsssp::values_d = values_d;

        tjnsssp::cur_modified_size_d = cur_modified_size_d;
        tjnsssp::cur_modified_size_d[0] = 1;

        tjnsssp::cur_modified_d = cur_modified_d;
        tjnsssp::cur_modified_d[0] = FLAGS_sssp_source;
        
        tjnsssp::num = num;
        tjnsssp::is_modified_d = is_modified_d;

        tjnsssp::node_type_d = node_type_d;
        tjnsssp::sem = sem_d;
        tjnsssp::sem[0] = 0;
        

        // hipFree(tjnsssp::next_modified_size);
    }

    void g_function(unsigned int *cur_modified_size_h, unsigned int num){
        dim3 block(512);
        dim3 grid((cur_modified_size_h[0] - 1) / block.x + 1);

        // printf("cur modified size is %d", cur_modified_size_h[0]);
        g_function_real<<<grid, block>>>();
        

        hipDeviceSynchronize();
        // unsigned int *next_modified_size_h = (unsigned int *)malloc(sizeof(unsigned int) * 1);
        // unsigned int *next_modified_size_d; hipMalloc(&next_modified_size_d, sizeof(unsigned int) * 1);


        dim3 block1(512);
        dim3 grid1((num-1) / block1.x + 1);
        setNextSize<<<grid1, block1>>>();
        hipDeviceSynchronize();

        swap(num);

        clear(num);
    }

    void g_function_compr(unsigned int *cur_modified_size_h, unsigned int cpr_num){
        dim3 block(512);
        dim3 grid((cur_modified_size_h[0] - 1) / block.x + 1);
        g_function_compr_real<<<grid, block>>>();
        hipDeviceSynchronize();
        // swap();
    }

    __global__
    void g_function_real(){
        
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        
        if(index < cur_modified_size_d[0]){
            sssp_Ingress(index);
        }
    }

    __global__
    void g_function_compr_real(){

        int index = threadIdx.x + blockIdx.x * blockDim.x;
        if(index < cur_modified_size_d[0]){
            unsigned int cur_modified_node = cur_modified_d[index];
            switch(node_type_d[cur_modified_node]){
                case 0:
                    {
                        sssp_nodeTypeZeroAndOne(index);
                    }
                case 1:
                    {
                        sssp_nodeTypeZeroAndOne(index);
                    }
                case 2:
                    {
                        sssp_nodeTypeTwo(index);
                    }
                case 3:
                    {
                        sssp_nodeTypeThree(index);
                    }
            }
        }
    }

    
    void swap(unsigned int num){
        hipEvent_t startCuda, stopCuda;  //declare
        hipEventCreate(&startCuda);      //set up 
        hipEventCreate(&stopCuda);       //set up
        hipEventRecord(startCuda,0);    //start
        
        unsigned int *next_modified_allsize_h = (unsigned int *)malloc(sizeof(unsigned int) * 1);
        unsigned int *next_modified_allsize_temp;
        hipMalloc(&next_modified_allsize_temp, sizeof(unsigned int)*1);

        getAllSize<<<1, 1>>>(next_modified_allsize_temp);
        hipMemcpy(next_modified_allsize_h, next_modified_allsize_temp, sizeof(unsigned int) * 1, hipMemcpyDeviceToHost);

        dim3 block(512);
        dim3 grid((num - 1) / block.x + 1);

        swap_real<<<grid, block>>>();
        hipEventRecord(stopCuda,0);     //finish
        hipEventSynchronize(stopCuda);
        hipDeviceSynchronize();
        float eTime;
        hipEventElapsedTime(&eTime, startCuda, stopCuda);  
        //eTime = stoptime - starttime
        printf("time is %f\n",eTime);

        hipFree(next_modified_allsize_temp);
        free(next_modified_allsize_h);
    }

    __global__
    void swap_real(){
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        if(index < num){
            if(is_modified_d[index]){
                
                // acquire_semaphore();
                for (int i = 0; i < 32; i++) {
                    // Check if it is this thread's turn
                    if (index % 32 != i)
                        continue;

                    // Lock
                    while (atomicExch(sem, 1) == 1)
                        ;
                    // Work
                    cur_modified_d[curpos] = index;
                    curpos++;
                    // Unlock
                    *sem = 0;
                }
                // while(true){
                //     // printf("sem = %d",sem);
                //     if(atomicExch(sem,1) != 1){
                //         printf("sem = %d",*sem);
                //         cur_modified_d[curpos] = index;
                //         curpos++;
                //         printf("sem = %d",*sem);
                //         *sem = 0;printf("sem = %d",*sem);
                //         break;
                //     }
                // }
                
                // cur_modified_d[curpos] = index;
                // curpos++;
                // printf("111");
                // sem = 0;
                // __syncthreads();
            }
        }
        
        // if(index < next_modified_allsize_d){
        //     int number = index + 1;
        //     for(unsigned int i = 0; i < num; i++){
        //         if(is_modified_d[i]){
        //             number--;
        //         }
        //         if(number == 0){
        //             cur_modified_d[index] = i;
        //             break;
        //         }
        //     }
        //     cur_modified_size_d[0] = next_modified_allsize_d;
        // }
        if(next_modified_allsize_d == 0 && index == 0){
            cur_modified_size_d[0] = 0;
            int maxid = 0;
            int maxvalue = 0;
            for(int i=0;i<3000000;i++){
                if(values_d[i] !=  2147483647 && values_d[i] > maxvalue){
                    maxvalue = values_d[i];
                    maxid = i;
                }
            }
            printf("max values[%d] is %d",maxid,values_d[maxid]);
            printf("-------------------------------end-------------------------------");
        }
    }

    void clear(unsigned int num){
        dim3 block(512);
        dim3 grid((num-1) / block.x + 1);
        clear_real<<<grid, block>>>();
    }

    __global__
    void clear_real(){

        int index = threadIdx.x + blockIdx.x * blockDim.x;
        

        if(index < num)
            is_modified_d[index] = 0;

        next_modified_allsize_d = 0;
        curpos = 0;

    }

    __global__
    void setNextSize(){
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        if(index < num && is_modified_d[index]){
            atomicAdd(&next_modified_allsize_d, 1);
        }
    }

    __global__
    void getAllSize(unsigned int *next_modified_allsize_temp){
        next_modified_allsize_temp[0] = next_modified_allsize_d;
        cur_modified_size_d[0] = next_modified_allsize_d;
    }

    __device__ 
    void acquire_semaphore(){
        // int index = threadIdx.x + blockIdx.x * blockDim.x;
        // int temp = atomicCAS(&sem, 0, 1);
        // printf("temp hhh is %d",temp);
        // if(temp == 0){
        //     if(index == 2)printf("no");
        //     return ;
        // }
        // while (temp != 0){
        //     temp = atomicCAS(&sem, 0, 1);
        //     // printf("temp while is %d index is %d sem is %d\n",temp,index,sem);
        // }
        
    }

    __device__ 
    void release_sem(){
        // sem = 0;
        // printf("sem is 0");
    }

    __device__
    void sssp_Ingress(int index){
        unsigned int cur_modified_node = cur_modified_d[index];
        if(values_d[cur_modified_node] > deltas_d[cur_modified_node]){
            // values_d[cur_modified_node] = deltas_d[cur_modified_node];
            atomicExch(&values_d[cur_modified_node], deltas_d[cur_modified_node]);
            for(unsigned int i = cur_oeoff_d[cur_modified_node]; i < size_oe_d[cur_modified_node] + cur_oeoff_d[cur_modified_node]; i++){
                unsigned int dist_node = oeoffset_d[i];
                // if(cur_modified_node == 9){
                //     printf("9 dist is %d",dist_node);
                // }
                // int new_dist = oe_edata_d[i] + deltas_d[cur_modified_node];//权重图
                int new_dist = 1 + deltas_d[cur_modified_node];//无权测试
                if(new_dist < deltas_d[dist_node]){
                    atomicMin(&deltas_d[dist_node], new_dist);
                    atomicExch(&is_modified_d[dist_node], 1);
                }
            }
        }
        __syncthreads();
        __threadfence();
    }

    __device__
    void sssp_nodeTypeZeroAndOne(int index){
        unsigned int cur_modified_node = cur_modified_d[index];
        if(values_d[cur_modified_node] > deltas_d[cur_modified_node]){
            // values_d[cur_modified_node] = deltas_d[cur_modified_node];
            atomicExch(&values_d[cur_modified_node], deltas_d[cur_modified_node]);
            for(unsigned int i = cur_iboff_d[cur_modified_node]; i < size_ib_d[cur_modified_node] + cur_iboff_d[cur_modified_node]; i++){
                unsigned int dist_node = iboffset_d[i];
                // if(cur_modified_node == 9){
                //     printf("9 dist is %d",dist_node);
                // }
                // int new_dist = ib_edata_d[i] + deltas_d[cur_modified_node];//权重图
                int new_dist = 1 + deltas_d[cur_modified_node];//无权测试
                if(new_dist < deltas_d[dist_node]){
                    atomicMin(&deltas_d[dist_node], new_dist);
                    atomicExch(&is_modified_d[dist_node], 1);
                }
            }
        }
    }

    __device__
    void sssp_nodeTypeTwo(int index){
        unsigned int cur_modified_node = cur_modified_d[index];
        if(values_d[cur_modified_node] > deltas_d[cur_modified_node]){
            // values_d[cur_modified_node] = deltas_d[cur_modified_node];
            atomicExch(&values_d[cur_modified_node], deltas_d[cur_modified_node]);
            for(unsigned int i = cur_isoff_d[cur_modified_node]; i < size_is_d[cur_modified_node] + cur_isoff_d[cur_modified_node]; i++){
                unsigned int dist_node = isoffset_d[i];
                // if(cur_modified_node == 9){
                //     printf("9 dist is %d",dist_node);
                // }
                // int new_dist = is_edata_d[i] + deltas_d[cur_modified_node];//权重图
                int new_dist = 1 + deltas_d[cur_modified_node];//无权测试
                if(new_dist < deltas_d[dist_node]){
                    atomicMin(&deltas_d[dist_node], new_dist);
                    atomicExch(&is_modified_d[dist_node], 1);
                }
            }
        }
    }

    __device__
    void sssp_nodeTypeThree(int index){
        unsigned int cur_modified_node = cur_modified_d[index];
        if(values_d[cur_modified_node] > deltas_d[cur_modified_node]){
            // values_d[cur_modified_node] = deltas_d[cur_modified_node];
            atomicExch(&values_d[cur_modified_node], deltas_d[cur_modified_node]);
            //第一阶段
            for(unsigned int i = cur_iboff_d[cur_modified_node]; i < size_ib_d[cur_modified_node] + cur_iboff_d[cur_modified_node]; i++){
                unsigned int dist_node = iboffset_d[i];
                // if(cur_modified_node == 9){
                //     printf("9 dist is %d",dist_node);
                // }
                // int new_dist = ib_edata_d[i] + deltas_d[cur_modified_node];//权重图
                int new_dist = 1 + deltas_d[cur_modified_node];//无权测试
                if(new_dist < deltas_d[dist_node]){
                    atomicMin(&deltas_d[dist_node], new_dist);
                    atomicExch(&is_modified_d[dist_node], 1);
                }
            }
            //第二阶段
            for(unsigned int i = cur_isoff_d[cur_modified_node]; i < size_is_d[cur_modified_node] + cur_isoff_d[cur_modified_node]; i++){
                unsigned int dist_node = isoffset_d[i];
                // if(cur_modified_node == 9){
                //     printf("9 dist is %d",dist_node);
                // }
                // int new_dist = ib_edata_d[i] + deltas_d[cur_modified_node];//权重图
                int new_dist = 1 + deltas_d[cur_modified_node];//无权测试
                if(new_dist < deltas_d[dist_node]){
                    atomicMin(&deltas_d[dist_node], new_dist);
                    atomicExch(&is_modified_d[dist_node], 1);
                }
            }
        }
    }

}